#include "hip/hip_runtime.h"
#include "Life_cuda.h"

__global__ void simpleLifeKernel(int width, int height, const uint8_t* grid, uint8_t* new_grid, int iteration) {
    int worldSize = width * height;
 
    for (int i = 0; i < iteration; i++)
        for (int cellId = __mul24(blockIdx.x, blockDim.x) + threadIdx.x; cellId < worldSize; cellId += blockDim.x * gridDim.x) {
            int x = cellId % width;
            int yAbs = cellId - x;
            int xLeft = (x + width - 1) % width;
            int xRight = (x + 1) % width;
            int yAbsUp = (yAbs + worldSize - width) % worldSize;
            int yAbsDown = (yAbs + width) % worldSize;
        
            int aliveCells = grid[xLeft + yAbsUp] + grid[x + yAbsUp]
            + grid[xRight + yAbsUp] + grid[xLeft + yAbs] + grid[xRight + yAbs]
            + grid[xLeft + yAbsDown] + grid[x + yAbsDown] + grid[xRight + yAbsDown];
        
            new_grid[x + yAbs] = aliveCells == 3 || (aliveCells == 2 && grid[x + yAbs]) ? 1 : 0;
        }
}

__global__ void bitLifeKernel(int width, int height, uint8_t* grid, uint8_t* new_grid, int iteration) {
    int x = blockIdx.x * blockDim.x + threadIdx.x + 1;
    int y = blockIdx.y * blockDim.y + threadIdx.y + 1;

    for (int i = 0; i < iteration; i++)
    {
        if (x <= width && y <= height) {
            for (int i = 0; i < 8; ++i) {
                uint8_t data0 = grid[(x - 1) * width + (y - 1)];
                uint8_t data1 = grid[(x    ) * width + (y - 1)];
                uint8_t data2 = grid[(x + 1) * width + (y - 1)];

                data0 <<= 1;
                data1 <<= 1;
                data2 <<= 1;

                uint8_t aliveCells = (data0 & 0x14) + (data1 & 0x14) + (data2 & 0x14);
                aliveCells >>= 2;

                int index = x * width + y;

                new_grid[index] = 0;
                if (aliveCells == 3) {
                    new_grid[index] = 1;
                } else if (aliveCells == 2 && grid[index] == 1) {
                    new_grid[index] = 1;
                }

                x = (x + 1) % width;
                y = (y + 1) % height;
            }
        }    
    }
}


__global__ void kernelUpdate(int width, int height, uint8_t* grid, uint8_t* new_grid, int iteration) {
    int x = blockIdx.x * blockDim.x + threadIdx.x + 1;
    int y = blockIdx.y * blockDim.y + threadIdx.y + 1;

    for (int i = 0; i < iteration; i++)
    {
        if (x <= width && y <= height) {
            // int m = getNeighbors(x, y, 1);
            int m = 0;
            m += grid[(x - 1)*width + (y    )];
            m += grid[(x    )*width + (y - 1)];
            m += grid[(x - 1)*width + (y - 1)];
            m += grid[(x + 1)*width + (y    )];
            m += grid[(x    )*width + (y + 1)];
            m += grid[(x + 1)*width + (y + 1)];
            m += grid[(x + 1)*width + (y - 1)];
            m += grid[(x - 1)*width + (y + 1)];
            
            int index = x*width + y;
            new_grid[index] = 0;
            if (m == 3) new_grid[index] = 1;          
            if (m == 2) new_grid[index] = grid[index];
        }
    }

}

void Life_cuda::update(int iteration)
{
	dim3 blockDim(5, 5);
    dim3 gridDim(width / blockDim.x, height / blockDim.y);

    uint8_t* d_grid;
	uint8_t* d_new_grid;

    // Allocate memory
    hipMalloc((void**)&d_grid, worldSize * sizeof(uint8_t));
    hipMalloc((void**)&d_new_grid, worldSize * sizeof(uint8_t));

    hipMemcpy(d_grid, grid, worldSize * sizeof(uint8_t), hipMemcpyHostToDevice);

    kernelUpdate<<<gridDim, blockDim>>>(width, height, d_grid, d_new_grid, iteration);
    // simpleLifeKernel<<<gridDim, blockDim>>>(width, height, d_grid, d_new_grid, iteration);
    // bitLifeKernel<<<gridDim, blockDim>>>(width, height, d_grid, d_new_grid, iteration);

    hipMemcpy(new_grid, d_new_grid, worldSize * sizeof(uint8_t), hipMemcpyDeviceToHost);

    // Free allocated memory
    hipFree(d_grid);
    hipFree(d_new_grid);

    swapGrids();
}