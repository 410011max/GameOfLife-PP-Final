#include "hip/hip_runtime.h"
#include "Life3d.h"

namespace life3
{
	__global__ void kernelUpdate(int width, int height, int depth, short* grid, short* new_grid)
    {
        int x = blockIdx.x * blockDim.x + threadIdx.x + 1;
        int y = blockIdx.y * blockDim.y + threadIdx.y + 1;
        int z = blockIdx.z * blockDim.z + threadIdx.z + 1;

        if (x <= width && y <= height && z <= depth)
        {
            // int m = getNeighbors(x, y, z);
            int m = 0;
		
			if (x > 0 && (x < width - 1) && y > 0 && (y < height - 1) && z > 0 && (z < depth - 1)) {
				m += grid[(y + 1) + x*width + z*width*height];
				m += grid[(y + 1) + x*width + (z+1)*width*height];
				m += grid[(y + 1) + x*width + (z-1)*width*height];

				m += grid[(y + 1) + (x + 1)*width + (z + 1)*width*height];
				m += grid[(y + 1) + (x + 1)*width + (z)*width*height];
				m += grid[(y + 1) + (x + 1)*width + (z - 1)*width*height];
				
				m += grid[(y + 1) + (x - 1)*width + z*width*height];
				m += grid[(y + 1) + (x - 1)*width + (z + 1)*width*height];
				m += grid[(y + 1) + (x - 1)*width + (z - 1)*width*height];
				//
				m += grid[(y - 1) + x*width + z*width*height];
				m += grid[(y - 1) + x*width + (z + 1)*width*height];
				m += grid[(y - 1) + x*width + (z - 1)*width*height];

				m += grid[(y - 1) + (x + 1)*width + (z + 1)*width*height];
				m += grid[(y - 1) + (x + 1)*width + (z)*width*height];
				m += grid[(y - 1) + (x + 1)*width + (z - 1)*width*height];

				m += grid[(y - 1) + (x - 1)*width + z*width*height];
				m += grid[(y - 1) + (x - 1)*width + (z + 1)*width*height];
				m += grid[(y - 1) + (x - 1)*width + (z - 1)*width*height];
				//
				m += grid[(y) + x*width + (z + 1)*width*height];
				m += grid[(y) + x*width + (z - 1)*width*height];

				m += grid[(y) + (x + 1)*width + (z + 1)*width*height];
				m += grid[(y) + (x + 1)*width + (z)*width*height];
				m += grid[(y) + (x + 1)*width + (z - 1)*width*height];

				m += grid[(y) + (x - 1)*width + z*width*height];
				m += grid[(y) + (x - 1)*width + (z + 1)*width*height];
				m += grid[(y) + (x - 1)*width + (z - 1)*width*height];
			}

			int index = z * width * height + y * width + x;
            new_grid[index] = 0;
            if (m < 3 || m > 5) 	new_grid[index] = 0;			// setNewLife(i, j, k, 0);
            if (m == 4 || m == 5) 	new_grid[index] = grid[index];	// setNewLife(i, j, k, getLifeform(i, j, k));
            if (m == 3) 			new_grid[index] = 1;			// setNewLife(i, j, k, 1);
        }
    }

    void Life3d::update()
    {
        dim3 blockDim(5, 5, 5);
        dim3 gridDim(width / blockDim.x, height / blockDim.y, depth / blockDim.z);

        short* d_grid;
        short* d_new_grid;

		// Allocate memory
        hipMalloc((void**)&d_grid, worldSize * sizeof(short));
        hipMalloc((void**)&d_new_grid, worldSize * sizeof(short));

        hipMemcpy(d_grid, grid, worldSize * sizeof(short), hipMemcpyHostToDevice);

        kernelUpdate<<<gridDim, blockDim>>>(width, height, depth, d_grid, d_new_grid);

        hipMemcpy(new_grid, d_new_grid, worldSize * sizeof(short), hipMemcpyDeviceToHost);

		// Free allocated memory
        hipFree(d_grid);
        hipFree(d_new_grid);

        swapGrids();
    }
}
