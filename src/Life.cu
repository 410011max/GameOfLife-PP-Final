#include "hip/hip_runtime.h"
#include "Life.h"


__global__ void kernelUpdate(int width, int height, short* grid, short* new_grid) {
    int x = blockIdx.x * blockDim.x + threadIdx.x + 1;
    int y = blockIdx.y * blockDim.y + threadIdx.y + 1;
    if (x <= width && y <= height) {
        // int m = getNeighbors(x, y, 1);
        int m = 0;
        m += grid[(x - 1)*width + (y    )];
        m += grid[(x    )*width + (y - 1)];
        m += grid[(x - 1)*width + (y - 1)];
        m += grid[(x + 1)*width + (y    )];
        m += grid[(x    )*width + (y + 1)];
        m += grid[(x + 1)*width + (y + 1)];
        m += grid[(x + 1)*width + (y - 1)];
        m += grid[(x - 1)*width + (y + 1)];
        
        int index = x*width + y;
        new_grid[index] = 0;
        if (m == 3)             new_grid[index] = 1;          
        if (m == 2)             new_grid[index] = grid[index];
    }
}

void Life::update()
{
	dim3 blockDim(5, 5);
    dim3 gridDim(width / blockDim.x, height / blockDim.y);

    short* d_grid;
	short* d_new_grid;

    // Allocate memory
    hipMalloc((void**)&d_grid, worldSize * sizeof(short));
    hipMalloc((void**)&d_new_grid, worldSize * sizeof(short));

    hipMemcpy(d_grid, grid, worldSize * sizeof(short), hipMemcpyHostToDevice);

    kernelUpdate<<<gridDim, blockDim>>>(width, height, d_grid, d_new_grid);

    hipMemcpy(new_grid, d_new_grid, worldSize * sizeof(short), hipMemcpyDeviceToHost);

    // Free allocated memory
    hipFree(d_grid);
    hipFree(d_new_grid);

    swapGrids();
}