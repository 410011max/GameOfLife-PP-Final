#include "hip/hip_runtime.h"
#include "Life.h"


__global__ void kernelUpdate(int width, int height, short* grid, short* new_grid) {
    int x = blockIdx.x * blockDim.x + threadIdx.x + 1;
    int y = blockIdx.y * blockDim.y + threadIdx.y + 1;

    if (x <= width && y <= height) {
        // int m = getNeighbors(x, y, 1);
        int m = 0;
        m += grid[(x - 1)*width + (y    )];
        m += grid[(x    )*width + (y - 1)];
        m += grid[(x - 1)*width + (y - 1)];
        m += grid[(x + 1)*width + (y    )];
        m += grid[(x    )*width + (y + 1)];
        m += grid[(x + 1)*width + (y + 1)];
        m += grid[(x + 1)*width + (y - 1)];
        m += grid[(x - 1)*width + (y + 1)];
        
        if (m == 3)
            new_grid[x*width + y] = 1;
            // setNewLife(x, y, 1);
        if (m == 2)
            new_grid[x*width + y] = grid[x*width + y];
            // setNewLife(x, y, getLifeform(x, y));
        if (m != 3 && m != 2)
            new_grid[x*width + y] = 0;
            // setNewLife(x, y, 0);
    }
}

void Life::update()
{
	dim3 blockDim(5, 5);
    dim3 gridDim(width / blockDim.x, height / blockDim.y);

    short* d_grid;
	short* d_new_grid;

    // Allocate memory
    hipHostRegister(grid, worldSize*sizeof(short), hipHostRegisterMapped);
    hipHostGetDevicePointer(&d_grid, grid, 0);
    hipHostRegister(new_grid, worldSize*sizeof(short), hipHostRegisterMapped);
    hipHostGetDevicePointer(&d_new_grid, new_grid, 0);

    kernelUpdate<<<gridDim, blockDim>>>(width, height, d_grid, d_new_grid);

    hipDeviceSynchronize();
    // Free allocated memory
    hipHostUnregister(grid);
    hipHostUnregister(new_grid);

	swapGrids();
}